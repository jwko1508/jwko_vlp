#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <time.h>
#include <math.h>

#define point_size 30000
static int *d_VMap;
static int *d_RMap;

static float *d_point_x = new float[point_size];
static float *d_point_y = new float[point_size];
static int *d_point_i = new int[point_size];
static double *d_w = new double[point_size];
static float *d_transed_point_x = new float[point_size];
static float *d_transed_point_y = new float[point_size];

__global__ void MeasInRMap(float *d_point_x, float *d_point_y, int *d_point_i, int *d_Map, float Map_resolution,
                           unsigned int Map_width, unsigned int Map_height, double Map_origin_x, double Map_origin_y, double *d_w)
{
    int tid = blockIdx.x;
    int xIndex, yIndex;
    int mapmeas;

    double resolutionInverse = 1/Map_resolution;
    if(tid < point_size){
        xIndex = (int)((d_point_x[tid] - Map_origin_x)*resolutionInverse);
        yIndex = (int)((d_point_y[tid] - Map_origin_y)*resolutionInverse);

        if(xIndex < Map_width && yIndex < Map_height){
            int mapIndex = Map_width*yIndex+xIndex;
            mapmeas = d_Map[mapIndex];
            if(mapmeas <0)
                mapmeas +=256;

            if(mapmeas > 100)
                d_w[tid] = 1;
            else
                d_w[tid] = 0;

            if(d_w[tid]>100)
                printf("road weight : %f\n",d_w[tid]);
        }
        else{
            d_w[tid]=0;
            printf("Out of RMap size!!!!!!!!!\n");
        }

    }
    else
        printf("Out of Road Point size!!!!!!!\n");

    }

__global__ void MeasInVMap(float *d_point_x, float *d_point_y, int *d_point_i, int *d_Map, float Map_resolution,
                          unsigned int Map_width, unsigned int Map_height, double Map_origin_x, double Map_origin_y, double *d_w)
{
    int tid = blockIdx.x;
    int xIndex, yIndex;
    int mapmeas;

    double resolutionInverse = 1/Map_resolution;
    if(tid<point_size){
        xIndex = (int)((d_point_x[tid] - Map_origin_x)*resolutionInverse);
        yIndex = (int)((d_point_y[tid] - Map_origin_y)*resolutionInverse);

        if(xIndex < Map_width && yIndex < Map_height){
            int mapIndex = Map_width*yIndex+xIndex;
            mapmeas = d_Map[mapIndex];
            if(mapmeas < 0)
                mapmeas +=256;

            int b1=0, b2=0;
            b1 = mapmeas&d_point_i[tid];
            int shBits;
            for (shBits=0; b1!=0;shBits++){
                b1 = b1 & (b1 -1);
            }

            // b2 = shBits*shBits;
            
            // if(mapmeas == d_point_i[tid] && b2!=1)
            //     b2=b2*2;

            // d_w[tid] = b2;
            d_w[tid] = shBits;
        }
        else{
printf("VMap%d\t%d\t%d\t%d\n",xIndex,Map_width,yIndex,Map_height);
            d_w[tid] = 0;
            printf("Out of VMap size!!!!!!!!!\n");
        }
    }
    else
        printf("Out of Vertical Point size!!!!!!!!!\n");

}

__global__ void Transformcuda(float *d_trans_point_x, float *d_trans_point_y, float *d_transed_point_x, float *d_transed_point_y, float Tx, float Ty, float theta)
{
    int tid = blockIdx.x;

    d_transed_point_x[tid] = d_trans_point_x[tid]*cos(theta) - d_trans_point_y[tid]*sin(theta) + Tx;
    d_transed_point_y[tid] = d_trans_point_x[tid]*sin(theta) + d_trans_point_y[tid]*cos(theta) + Ty;
}

double *MeasInMapCUDA(int N, float *point_x, float *point_y , int *point_i, int *Map, float Map_resolution,
                      unsigned int Map_width, unsigned int Map_height, double Map_origin_x, double Map_origin_y, float Tx, float Ty, float theta, double *w, std::string type)
{
    // Device copies of three inputs and output, size of allocated memory, num of threads and blocks
    hipMemcpy(d_point_x,point_x,N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_point_y,point_y,N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_point_i,point_i,N*sizeof(int), hipMemcpyHostToDevice);

    Transformcuda<<<N,1>>>(d_point_x, d_point_y, d_transed_point_x, d_transed_point_y, Tx, Ty, theta);
    if(type == "vertical")
        MeasInVMap<<<N,1>>>(d_transed_point_x, d_transed_point_y, d_point_i, d_VMap, Map_resolution, Map_width, Map_height, Map_origin_x, Map_origin_y, d_w);
    else if(type == "road")
        MeasInRMap<<<N,1>>>(d_transed_point_x, d_transed_point_y, d_point_i, d_RMap, Map_resolution, Map_width, Map_height, Map_origin_x, Map_origin_y, d_w);

    hipMemcpy(w, d_w, N*sizeof(double), hipMemcpyDeviceToHost);

    return w;
}

void CopyVMapCUDA(int *Map, unsigned int Map_width, unsigned int Map_height){
    d_VMap = new int[Map_width*Map_height];

    hipMalloc((void **)&d_VMap, Map_width*Map_height*sizeof(int));
    hipMemcpy(d_VMap, Map, Map_width*Map_height*sizeof(int), hipMemcpyHostToDevice);


}
void CopyRMapCUDA(int *Map, unsigned int Map_width, unsigned int Map_height) {
    d_RMap = new int[Map_width*Map_height];

    hipMalloc((void **)&d_RMap, Map_width * Map_height * sizeof(int));
    hipMemcpy(d_RMap, Map, Map_width * Map_height * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_point_x, point_size*sizeof(float));
    hipMalloc((void **)&d_point_y, point_size*sizeof(float));
    hipMalloc((void **)&d_point_i, point_size*sizeof(int));

    hipMalloc((void **)&d_transed_point_x, point_size*sizeof(float));
    hipMalloc((void **)&d_transed_point_y, point_size*sizeof(float));

    hipMalloc((void **)&d_w, point_size*sizeof(double));
}


void CUDAFree(){
    hipFree(&d_VMap); hipFree(&d_RMap); hipFree(&d_point_x); hipFree(&d_point_y); hipFree(&d_point_i); hipFree(&d_w);
    hipFree(&d_transed_point_x);hipFree(&d_transed_point_y);

}